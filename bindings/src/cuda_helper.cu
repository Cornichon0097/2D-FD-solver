#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include "cuda_helper.h"

#define N_ELEM(n) ((n) * (n))
#define MAT_SIZE(n) (N_ELEM(n) * sizeof(double))

#define INBOUND(idx, ncols) ((idx % ncols > 0) && (idx % ncols < ncols - 1) \
                             && (idx > ncols) && (idx < ncols * (ncols - 1)))

struct d_param {
        size_t ncols;
        double h_bar, m;
        double dx, dy, dt;
};

double *v0;
double *rpart, *ipart;
double *trpart, *tipart;
struct d_param *param;

size_t ncols = 0;

/**
 * \brief      { function_description }
 */
__global__ void ftcs(const double *const v0, const double *const rpart,
                     const double *const ipart, double *const trpart,
                     double *const tipart, const struct d_param *const p)
{
        int idx = blockDim.x * blockIdx.x + threadIdx.x;

        double const_dx = p->h_bar / (2 * p->m * p->dx * p->dx);
        double const_dy = p->h_bar / (2 * p->m * p->dy * p->dy);
        double potentiel;

        if (INBOUND(idx, p->ncols)) {
                potentiel = ((-1 / p->h_bar) * v0[idx])
                            - 2 * const_dx - 2 * const_dy;

                trpart[idx] = rpart[idx] - p->dt
                              * ((potentiel * ipart[idx])
                              + const_dx * (ipart[idx - p->ncols]
                                            + ipart[idx + p->ncols])
                              + const_dy * (ipart[idx - 1]
                                            + ipart[idx + 1]));

                tipart[idx] = ipart[idx] + p->dt
                              * ((potentiel * rpart[idx])
                              + const_dx * (rpart[idx - p->ncols]
                                            + rpart[idx + p->ncols])
                              + const_dy * (rpart[idx - 1]
                                            + rpart[idx + 1]));
    }
}

/**
 * \brief      Initializes the device matrix.
 *
 * \param[in]  h_data  The h data
 *
 * \return     { description_of_the_return_value }
 */
__host__ int init_device_matrix(const struct h_data *const h_data)
{
        hipError_t err = hipMalloc(&v0, MAT_SIZE(h_data->ncols));

        if (err != hipSuccess) {
                fprintf(stderr, "hipMalloc(): failed to allocate memory:"
                                "%s\n", hipGetErrorString(err));
                return -1;
        }

        err = hipMalloc(&rpart, MAT_SIZE(h_data->ncols));

        if (err != hipSuccess) {
                fprintf(stderr, "hipMalloc(): failed to allocate memory:"
                                "%s\n", hipGetErrorString(err));
                return -1;
        }

        err = hipMalloc(&ipart, MAT_SIZE(h_data->ncols));

        if (err != hipSuccess) {
                fprintf(stderr, "hipMalloc(): failed to allocate memory:"
                                "%s\n", hipGetErrorString(err));
                return -1;
        }

        err = hipMalloc(&trpart, MAT_SIZE(h_data->ncols));

        if (err != hipSuccess) {
                fprintf(stderr, "hipMalloc(): failed to allocate memory:"
                                "%s\n", hipGetErrorString(err));
                return -1;
        }

        err = hipMalloc(&tipart, MAT_SIZE(h_data->ncols));

        if (err != hipSuccess) {
                fprintf(stderr, "hipMalloc(): failed to allocate memory:"
                                "%s\n", hipGetErrorString(err));
                return -1;
        }

        err = hipMemcpy(v0, h_data->v0, MAT_SIZE(h_data->ncols),
                         hipMemcpyHostToDevice);

        if (err != hipSuccess) {
                fprintf(stderr, "hipMemcpy(): failed to copy host memory: "
                                "%s\n", hipGetErrorString(err));
                return -1;
        }

        err = hipMemcpy(rpart, h_data->rpart, MAT_SIZE(h_data->ncols),
                         hipMemcpyHostToDevice);

        if (err != hipSuccess) {
                fprintf(stderr, "hipMemcpy(): failed to copy host memory: "
                                "%s\n", hipGetErrorString(err));
                return -1;
        }

        err = hipMemcpy(ipart, h_data->ipart, MAT_SIZE(h_data->ncols),
                         hipMemcpyHostToDevice);

        if (err != hipSuccess) {
                fprintf(stderr, "hipMemcpy(): failed to copy host memory: "
                                "%s\n", hipGetErrorString(err));
                return -1;
        }

        return 0;
}

/**
 * \brief      Initializes the device memory.
 *
 * \param[in]  h_data  The h data
 *
 * \return     { description_of_the_return_value }
 */
__host__ int init_device_memory(const struct h_data *const h_data)
{
        hipError_t err;

        if (ncols != 0) {
                fprintf(stderr, "Device memory already initialized\n");
                return -1;
        }

        if (init_device_matrix(h_data) != 0)
                return -1;

        err = hipMalloc(&param, sizeof(struct d_param));

        if (err != hipSuccess) {
                fprintf(stderr, "hipMalloc(): failed to allocate memory:"
                                "%s\n", hipGetErrorString(err));
                return -1;
        }

        hipMemcpy(param, &(h_data->ncols), sizeof(struct d_param),
                   hipMemcpyHostToDevice);

        if (err != hipSuccess) {
                fprintf(stderr, "hipMemcpy(): failed to copy host memory: "
                                "%s\n", hipGetErrorString(err));
                return -1;
        }

        ncols = h_data->ncols;

        return 0;
}

/**
 * \brief      { function_description }
 *
 * \param[in]  scheme  The scheme
 *
 * \return     { description_of_the_return_value }
 */
__host__ int execute_kernel(const char scheme)
{
        hipError_t err;

        switch (scheme) {
        case 'f':
                ftcs<<<ncols, ncols>>>(v0, rpart, ipart, trpart, tipart, param);
                break;
        default:
                break;
        }

        err = hipGetLastError();

        if (err != hipSuccess) {
                fprintf(stderr, "Failed to launch kernel:"
                                "%s\n", hipGetErrorString(err));
                return -1;
        }

        hipDeviceSynchronize();

        err = hipMemcpy(rpart, trpart, MAT_SIZE(ncols),
                         hipMemcpyDeviceToDevice);

        if (err != hipSuccess) {
                fprintf(stderr, "hipMemcpy(): failed to copy host memory: "
                                "%s\n", hipGetErrorString(err));
                return -1;
        }

        err = hipMemcpy(ipart, tipart, MAT_SIZE(ncols),
                         hipMemcpyDeviceToDevice);

        if (err != hipSuccess) {
                fprintf(stderr, "hipMemcpy(): failed to copy host memory: "
                                "%s\n", hipGetErrorString(err));
                return -1;
        }

        return 0;
}

/**
 * \brief      Retrieves results.
 *
 * \param[out] res   The resource
 *
 * \return     The results.
 */
__host__ int retrieve_results(struct h_data *const res)
{
        hipError_t err;

        if (res == NULL) {
                fprintf(stderr, "Impossible to retrieve results in (null)\n");
                return -1;
        }

        err = hipMemcpy(res->rpart, rpart, MAT_SIZE(ncols),
                   hipMemcpyDeviceToHost);

        if (err != hipSuccess) {
                fprintf(stderr, "hipMemcpy(): failed to copy device memory: "
                                "%s\n", hipGetErrorString(err));
                return -1;
        }

        err = hipMemcpy(res->ipart, ipart, MAT_SIZE(ncols),
                   hipMemcpyDeviceToHost);

        if (err != hipSuccess) {
                fprintf(stderr, "hipMemcpy(): failed to copy device memory: "
                                "%s\n", hipGetErrorString(err));
                return -1;
        }

        return 0;
}

/**
 * \brief      { function_description }
 */
__host__ int clean_up_device(void)
{
        hipError_t err = hipDeviceReset();

        if (err != hipSuccess) {
                fprintf(stderr, "Failed to clean the device:"
                                " %s\n", hipGetErrorString(err));
                return -1;
        }

        return 0;
}
