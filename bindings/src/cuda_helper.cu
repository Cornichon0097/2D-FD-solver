#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include "cuda_helper.h"

#define N_ELEM(n) (n * n)
#define MAT_SIZE(n) (N_ELEM(n) * sizeof(double))

#define INBOUND(idx, ncols) ((idx % ncols > 0) && (idx % ncols < ncols - 1) \
                             && (idx > ncols) && (idx < ncols * (ncols - 1)))

struct d_data *d_data = NULL;

double *d_rpart, *d_ipart;

static size_t ncols = 0;

/**
 * \brief      { function_description }
 */
__global__ void ftcs(double *drp, double *dip, struct d_data *const d)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    double const_dx = d->h_bar / (2 * d->m * d->dx * d->dx);
    double const_dy = d->h_bar / (2 * d->m * d->dy * d->dy);
    double potentiel;

    if (INBOUND(idx, d->ncols)) {
        potentiel = ((-1 / d->h_bar) * d->v0[idx])
                    - 2 * const_dx - 2 * const_dy;

        drp[idx] = d->rpart[idx] - d->dt
                * ((potentiel * d->ipart[idx])
                + const_dx * (d->ipart[idx - d->ncols]
                        + d->ipart[idx + d->ncols])
                + const_dy * (d->ipart[idx - 1] + d->ipart[idx + 1]));

        dip[idx] = d->ipart[idx] + d->dt
                * ((potentiel * d->rpart[idx])
                + const_dx * (d->rpart[idx - d->ncols]
                        + d->rpart[idx + d->ncols])
                + const_dy * (d->rpart[idx - 1] + d->rpart[idx + 1]));
    }
}

/**
 * \brief      Initializes the device matrix.
 *
 * \param[in]  h_data  The h data
 *
 * \return     { description_of_the_return_value }
 */
__host__ int init_device_matrix(struct d_data *const th_data, const struct d_data *const h_data)
{
        hipError_t err = hipMalloc(&d_rpart, MAT_SIZE(ncols));

        if (err != hipSuccess) {
                fprintf(stderr, "hipMalloc(): failed to allocate memory:"
                                "%s\n", hipGetErrorString(err));
                return -1;
        }

        err = hipMalloc(&d_ipart, MAT_SIZE(ncols));

        if (err != hipSuccess) {
                fprintf(stderr, "hipMalloc(): failed to allocate memory:"
                                "%s\n", hipGetErrorString(err));
                return -1;
        }

        err = hipMalloc(&(th_data->v0), MAT_SIZE(ncols));

        if (err != hipSuccess) {
                fprintf(stderr, "hipMalloc(): failed to allocate memory:"
                                "%s\n", hipGetErrorString(err));
                return -1;
        }

        err = hipMemcpy(th_data->v0, h_data->v0, MAT_SIZE(ncols),
                         hipMemcpyHostToDevice);

        if (err != hipSuccess) {
                fprintf(stderr, "hipMemcpy(): failed to copy host memory: "
                                "%s\n", hipGetErrorString(err));
                return -1;
        }

        err = hipMalloc(&(th_data->rpart), MAT_SIZE(ncols));

        if (err != hipSuccess) {
                fprintf(stderr, "hipMalloc(): failed to allocate memory:"
                                "%s\n", hipGetErrorString(err));
                return -1;
        }

        err = hipMemcpy(th_data->rpart, h_data->rpart, MAT_SIZE(ncols),
                         hipMemcpyHostToDevice);

        if (err != hipSuccess) {
                fprintf(stderr, "hipMemcpy(): failed to copy host memory: "
                                "%s\n", hipGetErrorString(err));
                return -1;
        }

        err = hipMalloc(&(th_data->ipart), MAT_SIZE(ncols));

        if (err != hipSuccess) {
                fprintf(stderr, "hipMalloc(): failed to allocate memory:"
                                "%s\n", hipGetErrorString(err));
                return -1;
        }

        err = hipMemcpy(th_data->ipart, h_data->ipart, MAT_SIZE(ncols),
                         hipMemcpyHostToDevice);

        if (err != hipSuccess) {
                fprintf(stderr, "hipMemcpy(): failed to copy host memory: "
                                "%s\n", hipGetErrorString(err));
                return -1;
        }

        return 0;
}

/**
 * \brief      Initializes the device memory.
 *
 * \param[in]  h_data  The h data
 *
 * \return     { description_of_the_return_value }
 */
__host__ int init_device_memory(const struct d_data *const h_data)
{
        struct d_data tmp_h_data;

        if (ncols != 0) {
                fprintf(stderr, "Device memory already initialized\n");
                return -1;
        }

        ncols = h_data->ncols;

        if (init_device_matrix(&tmp_h_data, h_data) != 0)
                return -1;

        tmp_h_data.ncols = h_data->ncols;
        tmp_h_data.h_bar = h_data->h_bar;
        tmp_h_data.m     = h_data->m;
        tmp_h_data.dx    = h_data->dx;
        tmp_h_data.dy    = h_data->dy;
        tmp_h_data.dt    = h_data->dt;

        hipMalloc(&d_data, sizeof(struct d_data));
        hipMemcpy(d_data, &tmp_h_data, sizeof(struct d_data), hipMemcpyHostToDevice);

        return 0;
}

/**
 * \brief      { function_description }
 *
 * \param[in]  scheme  The scheme
 *
 * \return     { description_of_the_return_value }
 */
__host__ int execute_kernel(const char scheme)
{
        hipError_t err;

        switch (scheme) {
        case 'f':
                ftcs<<<1, ncols>>>(d_rpart, d_ipart, d_data);
                break;
        default:
                break;
        }

        err = hipGetLastError();

        if (err != hipSuccess) {
                fprintf(stderr, "Failed to launch kernel:"
                                "%s\n", hipGetErrorString(err));
                return -1;
        }

        hipDeviceSynchronize();

        return 0;
}

/**
 * \brief      Retrieves results.
 *
 * \param[out] res   The resource
 *
 * \return     The results.
 */
__host__ int retrieve_results(struct d_data *const res)
{
        struct d_data tmp_h_data;
        hipError_t err;

        if (res == NULL) {
                fprintf(stderr, "Impossible to retrieve results in (null)\n");
                return -1;
        }

        hipMemcpy(d_data, &tmp_h_data, sizeof(struct d_data), hipMemcpyDeviceToHost);

        err = hipMemcpy(&(tmp_h_data.rpart), res->rpart, MAT_SIZE(ncols),
                         hipMemcpyDeviceToHost);

        if (err != hipSuccess) {
                fprintf(stderr, "hipMemcpy(): failed to copy device memory: "
                                "%s\n", hipGetErrorString(err));
                return -1;
        }

        err = hipMemcpy(&(tmp_h_data.ipart), res->ipart, MAT_SIZE(ncols),
                         hipMemcpyDeviceToHost);

        if (err != hipSuccess) {
                fprintf(stderr, "hipMemcpy(): failed to copy device memory: "
                                "%s\n", hipGetErrorString(err));
                return -1;
        }

        return 0;
}

/**
 * \brief      { function_description }
 */
__host__ int clean_up_device(void)
{
        hipError_t err = hipDeviceReset();

        if (err != hipSuccess) {
                fprintf(stderr, "Failed to clean the device:"
                                " %s\n", hipGetErrorString(err));
                return -1;
        }

        return 0;
}
