#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include "cuda_helper.h"

#define N_ELEM(n) (n * n)
#define MAT_SIZE(n) (N_ELEM(n) * sizeof(double))

#define INBOUND(idx, ncols) ((idx % ncols > 0) && (idx % ncols < ncols - 1) \
                             && (idx > ncols) && (idx < ncols * (ncols - 1)))

struct d_param {
        size_t ncols;
        double h_bar, m;
        double dx, dy, dt;
};

struct d_data {
        double *v0;
        double *rpart, *ipart;
        double *trpart, *tipart;
        struct d_param *param;
};

struct d_data *d_data = NULL;
size_t ncols;

/**
 * \brief      { function_description }
 */
__global__ void ftcs(const double *const v0, const double *const rpart,
                     const double *const ipart, double *const trpart,
                     double *const tipart, const struct d_param *const p)
{
        int idx = blockDim.x * blockIdx.x + threadIdx.x;

        double const_dx = p->h_bar / (2 * p->m * p->dx * p->dx);
        double const_dy = p->h_bar / (2 * p->m * p->dy * p->dy);
        double potentiel;

        if (INBOUND(idx, p->ncols)) {
                potentiel = ((-1 / p->h_bar) * v0[idx])
                            - 2 * const_dx - 2 * const_dy;

                trpart[idx] = rpart[idx] - p->dt
                              * ((potentiel * ipart[idx])
                              + const_dx * (ipart[idx - p->ncols]
                                            + ipart[idx + p->ncols])
                              + const_dy * (ipart[idx - 1]
                                            + ipart[idx + 1]));

                tipart[idx] = ipart[idx] + p->dt
                              * ((potentiel * rpart[idx])
                              + const_dx * (rpart[idx - p->ncols]
                                            + rpart[idx + p->ncols])
                              + const_dy * (rpart[idx - 1]
                                            + rpart[idx + 1]));
    }
}

/**
 * \brief      Initializes the device matrix.
 *
 * \param[in]  h_data  The h data
 *
 * \return     { description_of_the_return_value }
 */
__host__ int init_device_matrix(const struct h_data *const h_data)
{
        hipError_t err = hipMalloc(&(d_data->v0), MAT_SIZE(h_data->ncols));

        if (err != hipSuccess) {
                fprintf(stderr, "hipMalloc(): failed to allocate memory:"
                                "%s\n", hipGetErrorString(err));
                return -1;
        }

        err = hipMalloc(&(d_data->rpart), MAT_SIZE(h_data->ncols));

        if (err != hipSuccess) {
                fprintf(stderr, "hipMalloc(): failed to allocate memory:"
                                "%s\n", hipGetErrorString(err));
                return -1;
        }

        err = hipMalloc(&(d_data->ipart), MAT_SIZE(h_data->ncols));

        if (err != hipSuccess) {
                fprintf(stderr, "hipMalloc(): failed to allocate memory:"
                                "%s\n", hipGetErrorString(err));
                return -1;
        }

        err = hipMalloc(&(d_data->trpart), MAT_SIZE(h_data->ncols));

        if (err != hipSuccess) {
                fprintf(stderr, "hipMalloc(): failed to allocate memory:"
                                "%s\n", hipGetErrorString(err));
                return -1;
        }

        err = hipMalloc(&(d_data->tipart), MAT_SIZE(h_data->ncols));

        if (err != hipSuccess) {
                fprintf(stderr, "hipMalloc(): failed to allocate memory:"
                                "%s\n", hipGetErrorString(err));
                return -1;
        }

        err = hipMemcpy(d_data->v0, h_data->v0, MAT_SIZE(h_data->ncols),
                         hipMemcpyHostToDevice);

        if (err != hipSuccess) {
                fprintf(stderr, "hipMemcpy(): failed to copy host memory: "
                                "%s\n", hipGetErrorString(err));
                return -1;
        }

        err = hipMemcpy(d_data->rpart, h_data->rpart, MAT_SIZE(h_data->ncols),
                         hipMemcpyHostToDevice);

        if (err != hipSuccess) {
                fprintf(stderr, "hipMemcpy(): failed to copy host memory: "
                                "%s\n", hipGetErrorString(err));
                return -1;
        }

        err = hipMemcpy(d_data->ipart, h_data->ipart, MAT_SIZE(h_data->ncols),
                         hipMemcpyHostToDevice);

        if (err != hipSuccess) {
                fprintf(stderr, "hipMemcpy(): failed to copy host memory: "
                                "%s\n", hipGetErrorString(err));
                return -1;
        }

        return 0;
}

/**
 * \brief      Initializes the device memory.
 *
 * \param[in]  h_data  The h data
 *
 * \return     { description_of_the_return_value }
 */
__host__ int init_device_memory(const struct h_data *const h_data)
{
        hipError_t err;

        if (d_data != NULL) {
                fprintf(stderr, "Device memory already initialized\n");
                return -1;
        }

        d_data = (struct d_data *) malloc(sizeof(struct d_data));

        if (d_data == NULL) {
                perror("malloc()");
                return -1;
        }

        if (init_device_matrix(h_data) != 0)
                return -1;

        err = hipMalloc(&(d_data->param), sizeof(struct d_param));

        if (err != hipSuccess) {
                fprintf(stderr, "hipMalloc(): failed to allocate memory:"
                                "%s\n", hipGetErrorString(err));
                return -1;
        }

        hipMemcpy(d_data->param, &(h_data->ncols), sizeof(struct d_param),
                   hipMemcpyHostToDevice);

        if (err != hipSuccess) {
                fprintf(stderr, "hipMemcpy(): failed to copy host memory: "
                                "%s\n", hipGetErrorString(err));
                return -1;
        }

        ncols = h_data->ncols;

        return 0;
}

/**
 * \brief      { function_description }
 *
 * \param[in]  scheme  The scheme
 *
 * \return     { description_of_the_return_value }
 */
__host__ int execute_kernel(const char scheme)
{
        hipError_t err;

        switch (scheme) {
        case 'f':
                ftcs<<<1, ncols>>>(d_data->v0, d_data->rpart, d_data->ipart,
                                   d_data->trpart, d_data->tipart,
                                   d_data->param);
                break;
        default:
                break;
        }

        err = hipGetLastError();

        if (err != hipSuccess) {
                fprintf(stderr, "Failed to launch kernel:"
                                "%s\n", hipGetErrorString(err));
                return -1;
        }

        hipDeviceSynchronize();

        err = hipMemcpy(d_data->rpart, d_data->trpart, MAT_SIZE(ncols),
                         hipMemcpyDeviceToDevice);

        if (err != hipSuccess) {
                fprintf(stderr, "hipMemcpy(): failed to copy host memory: "
                                "%s\n", hipGetErrorString(err));
                return -1;
        }

        err = hipMemcpy(d_data->ipart, d_data->tipart, MAT_SIZE(ncols),
                         hipMemcpyDeviceToDevice);

        if (err != hipSuccess) {
                fprintf(stderr, "hipMemcpy(): failed to copy host memory: "
                                "%s\n", hipGetErrorString(err));
                return -1;
        }

        return 0;
}

/**
 * \brief      Retrieves results.
 *
 * \param[out] res   The resource
 *
 * \return     The results.
 */
__host__ int retrieve_results(struct h_data *const res)
{
        hipError_t err;

        if (res == NULL) {
                fprintf(stderr, "Impossible to retrieve results in (null)\n");
                return -1;
        }

        err = hipMemcpy(res->rpart, d_data->rpart, MAT_SIZE(ncols),
                   hipMemcpyDeviceToHost);

        if (err != hipSuccess) {
                fprintf(stderr, "hipMemcpy(): failed to copy device memory: "
                                "%s\n", hipGetErrorString(err));
                return -1;
        }

        err = hipMemcpy(res->ipart, d_data->ipart, MAT_SIZE(ncols),
                   hipMemcpyDeviceToHost);

        if (err != hipSuccess) {
                fprintf(stderr, "hipMemcpy(): failed to copy device memory: "
                                "%s\n", hipGetErrorString(err));
                return -1;
        }

        return 0;
}

/**
 * \brief      { function_description }
 */
__host__ int clean_up_device(void)
{
        hipError_t err = hipDeviceReset();

        if (err != hipSuccess) {
                fprintf(stderr, "Failed to clean the device:"
                                " %s\n", hipGetErrorString(err));
                return -1;
        }

        return 0;
}
